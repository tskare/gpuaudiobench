#include "bench_base.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void GPUABenchmark::InitDefaultBuffers(
	std::vector<int> host_input_buffer_sizes_bytes,
	std::vector<int> host_output_buffer_sizes_bytes,
    std::vector<int> device_input_buffer_sizes_bytes,
    std::vector<int> device_output_buffer_sizes_bytes) {

    hipError_t err = hipSuccess;

    // TODO(travis): Support multiple buffers, or zero.
    // this->hIn = (float*)malloc(host_input_buffer_sizes_bytes[0]);
    // this->hOut = (float*)malloc(host_output_buffer_sizes_bytes[0]);
    // Allocate memory for the input and output buffers
    hipHostMalloc((void**)hIn, sizeof(float) * host_input_buffer_sizes_bytes[0]);
    hipHostMalloc((void**)hOut, sizeof(float) * host_output_buffer_sizes_bytes[0]);

    err = hipMalloc((void**)(this->dIn), device_input_buffer_sizes_bytes * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device input mem(error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)(this->dOut), device_output_buffer_sizes_bytes * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device input mem(error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
