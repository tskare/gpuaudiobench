#include "bench_datatransfer.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void SetupBenchmarkIO(float** h_inBuf, float** h_outBuf, float** d_inBuf, float** d_outBuf,
    const int IOTEST_INBUFCOUNT, const int IOTEST_OUTBUFCOUNT) {
    hipError_t err = hipSuccess;

    *h_inBuf = (float*)malloc(IOTEST_INBUFCOUNT * sizeof(float));
    *h_outBuf = (float*)malloc(IOTEST_OUTBUFCOUNT * sizeof(float));

    if (*h_inBuf == NULL || *h_outBuf == NULL) {
		fprintf(stderr, "Failed to allocate host buffers\n");
		exit(EXIT_FAILURE);
	}
    
    err = hipMalloc((void**)d_inBuf, IOTEST_INBUFCOUNT * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void**)d_outBuf, IOTEST_OUTBUFCOUNT * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Init host vector with random floats
    for (int i = 0; i < IOTEST_INBUFCOUNT; i++) {
        (*h_inBuf)[i] = rand() / (float)RAND_MAX;
    }
}