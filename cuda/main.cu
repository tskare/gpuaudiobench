#include "hip/hip_runtime.h"
/**
 * CUDA GPGPU Audio Microbenchmarks and I/O benchmarks suite.
 * See README in repository root for overall project details.
 */

 // The Visual Studio project and starter code was adapted from CUDA
 // toolkit examples, with the below license.

 /* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
  *
  * Redistribution and use in source and binary forms, with or without
  * modification, are permitted provided that the following conditions
  * are met:
  *  * Redistributions of source code must retain the above copyright
  *    notice, this list of conditions and the following disclaimer.
  *  * Redistributions in binary form must reproduce the above copyright
  *    notice, this list of conditions and the following disclaimer in the
  *    documentation and/or other materials provided with the distribution.
  *  * Neither the name of NVIDIA CORPORATION nor the names of its
  *    contributors may be used to endorse or promote products derived
  *    from this software without specific prior written permission.
  *
  * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
  * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
  * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
  * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
  * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
  * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
  * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
  * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
  */

  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// CUDA complex types, used for Modal filter bank trials.
#include <hip/hip_complex.h>

#include <iostream>
#include <vector>
#include <thread>

#include <hip/hip_runtime_api.h>

#include "globals.cuh"

// Benchmark includes
#include "bench_datatransfer.cuh"
#include "bench_modal.cuh"
//#include "bench_gainstats.cuh"
#include "benchmark_rndmem.cuh"

// See globals.cuh for cross-benchmark parameters.
// See individual .cuh files for benchmark-specific parameters.

// GPGPU Audio benchmark kernel:
// Test for data transfer, weighted
__global__ void DataTransferBenchmarkKernel(const float* bufIn, float* bufOut, int numElements) {
	// No-op, but keep this piece as the Metal impl performs the same implicitly.
	// It seems it is optimized out of the intermediate code however.
	int i = blockDim.x * blockIdx.x + threadIdx.x;
}

// Conv1D, time-domain implementation, using texture memory.
#include <hip/hip_texture_types.h>
__global__ void Conv1DTextureMemoryImplKernel( 
		const float* bufIn, float* bufOut, const hipTextureObject_t textureRefIRs, int irLen) {
	int whichThread = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < BUFSIZE; i++) {
		// Perform time-series convolution:
		float samp = 0.0f;
		for (int j = 0; j < irLen; j++) {
			// CLEANUP: experiment with iterating in other directions. Though
			// this seems to utilize caches as expected.
			samp += tex2D<float>(textureRefIRs, whichThread, j) * bufIn[whichThread * BUFSIZE + i - j];
		}
		bufOut[NTRACKS * i + whichThread] = samp;
	}
	// v2: We could tree-sum here to reduce I/O overhead for the benchmark.
}



// Kernel to compute basic gain 
__global__ void GainStatsKernel(const float* bufIn, float* bufOut, float* stats, int numElements) {
	int trackidx = blockDim.x * blockIdx.x + threadIdx.x;
	const int nstats = 2;
	float mean = 0.0f;
	for (int i = 0; i < BUFSIZE; i++) {
		float samp = bufIn[trackidx * BUFSIZE + i];
		mean += samp;
		bufOut[trackidx * BUFSIZE + i] = samp * 0.5f;
		// Max
		if (samp > stats[trackidx * (NTRACKS * nstats) + 0]) {
			stats[trackidx * (NTRACKS * nstats) + 0] = samp;
		}
	}
	mean /= BUFSIZE;
}

const char* benchmarkNames[] = {
	"RndMem",
	"GainStats",
	"Modal",
	"IO",
	"Conv1D_Beta"
};

static void printHelp() {
	printf("Usage: gpu-audio-benchmark [options]\n");
	printf("Options:\n");
	printf("  --help: Print this help message\n");
	printf("  --benchmark [X]: Run benchmark |benchmarkName|\n");
	printf("  --fs: set sampling rate\n");
	printf("  --bufferSize: set buffer size\n");
	printf("  --ntracks: set number of tracks\n");
	printf("\n");
	printf("Available benchmarks:\n");
	for (int i = 0; i < sizeof(benchmarkNames) / sizeof(benchmarkNames[0]); i++) {
		printf("  %s\n", benchmarkNames[i]);
	}
}

/**
 * Host main routine
 */
 // main() with argc and argv:
int main(int argc, char** argv) {
	printf("GPGPU Audio Benchmark\n");

	std::string whichBenchmark = "RndMem";

	// Skip first argument, the executable name.
	for (int i = 1; i < argc; i++) {
		bool hasNextParameter = i + 1 < argc;
		// printf("argv[%d]: %s\n", i, argv[i]);
		if (strcmp(argv[i], "--help") == 0) {
			printHelp();
			return 0;
		}
		if (strcmp(argv[i], "--benchmark") == 0) {
			if (!hasNextParameter) {
				printf("Error: --benchmark requires an argument\n");
				return 1;
			}
			// Arg validity is checked in init and run sections.
			whichBenchmark = std::string(argv[i + 1]);
			i++;
		} else if (strcmp(argv[i], "--fs") == 0) {
			if (!hasNextParameter) {
				printf("Error: --fs requires an argument\n");
				return 1;
			}
			FS = atoi(argv[i + 1]);
			i++;
		} else if (strcmp(argv[i], "--bufferSize") == 0) {
			if (!hasNextParameter) {
				printf("Error: --bufferSize requires an argument\n");
				return 1;
			}
			printf("Warning: currently transitioning from compile-time to runtime buffer size\n");
			//BUFSIZE = atoi(argv[i + 1]);
			i++;
		} else if (strcmp(argv[i], "--nTracks") == 0) {
			if (!hasNextParameter) {
				printf("Error: --nTracks requires an argument\n");
				return 1;
			}
			// NTRACKS = atoi(argv[i + 1]);
			printf("Warning: currently transitioning from compile-time to runtime nTracks\n");
			i++;
		} else if (strcmp(argv[i], "--nRuns") == 0) {
			if (!hasNextParameter) {
				printf("Error: --nRuns requires an argument\n");
				return 1;
			}
			NRUNS = atoi(argv[i + 1]);
			i++;
		}
		else {
			printf("Warning: Unparsed argument: %s\n", argv[i]);
		}
	}

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Intermediate data:
	// Moving from compile-time to runtime benchmark selection.
	// 

	// Buffer names, may be shared between benchmarks.
	// Individual benchmarks should allocate and free their own buffers.
	// Naming scheme: prefixed h_ for host, d_ for device.
	// Note CUDA provides shared memory, but we're using the more direct
	// and manual approach here, as the transfers are explicit.
	float* h_inBuf = nullptr;
	float* h_outBuf = nullptr;
	float* d_inBuf = nullptr;
	float* d_outBuf = nullptr;
	float* h_sampleMem = nullptr;
	float* d_sampleMem = nullptr;
	int* h_playheads = nullptr;
	int* d_playheads = nullptr;
	float* h_stats = nullptr;
	float* d_stats = nullptr;
	float* h_irBuf = nullptr;
	float* d_irBuf = nullptr;

	// Following is local data that should be moved to a data struct for
	// each corresponding benchmark:
	// RndMem
	float playheadsStart[NTRACKS];
	float playheadsEnd[NTRACKS];
	int minLoopLen = 1000;
	int maxLoopLen = 48000;
	int samplebufferEnd = kSampleMemNumElems - BUFSIZE;
	// Conv1D
	int irLen = 1024;
	hipArray_t cuArrayIRs = 0;
	hipTextureObject_t texObjIRs = 0;

	if (whichBenchmark == "IO") {
		SetupBenchmarkIO(&h_inBuf, &h_outBuf, &d_inBuf, &d_outBuf, IOTEST_INBUFCOUNT, IOTEST_OUTBUFCOUNT);
	}
	else if (whichBenchmark == "RndMem") {
		SetupBenchmarkRndMem(&h_sampleMem, &d_sampleMem,
			&h_playheads, &d_playheads,
			playheadsStart,
			playheadsEnd,
			minLoopLen,
			maxLoopLen,
			samplebufferEnd,
			&h_outBuf, &d_outBuf);
	}
	else if (whichBenchmark == "GainStats") {
		// TODO: Finish copying this out into its own file. Was here for debugging.
		h_inBuf = (float*)malloc(NTRACKS * BUFSIZE * sizeof(float));
		h_outBuf = (float*)malloc(NTRACKS * BUFSIZE * sizeof(float));
		constexpr int nstats = 4;
		d_stats = (float*)malloc(NTRACKS * nstats * sizeof(float));
		h_stats = (float*)malloc(NTRACKS * nstats * sizeof(float));

		err = hipMalloc((void**)&d_inBuf, NTRACKS * BUFSIZE * sizeof(float));
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		err = hipMalloc((void**)&d_outBuf, NTRACKS * BUFSIZE * sizeof(float));
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		// Init host vector with random floats
		for (int i = 0; i < NTRACKS * BUFSIZE; i++) {
			h_inBuf[i] = rand() / (float)RAND_MAX;
		}
	}
	else if (whichBenchmark == "Conv1D_Beta") {
		// Note for small numbers of constants you may wish to use
		// hipMemcpyToSymbol()
		// However, for NTRACKS each having an IR, this will not fit.
		// This benchmark thus tries to determine the speedup, if any, of using
		// texture memory.
		// We don't benefit from the extra interpolation and 2D spatial locality
		// the texture cache will provide, but might benefit from the cache
		// being separate from the global cache (IIUC).
		// In practice, at the time of this writing, performance seems identical
		// between using texture memory or not.

		// CLEANUP: Copy this into its own file.
		h_inBuf = (float*)malloc(NTRACKS * BUFSIZE * sizeof(float));
		h_outBuf = (float*)malloc(NTRACKS * BUFSIZE * sizeof(float));
		h_irBuf = (float*)malloc(NTRACKS * irLen * sizeof(float));

		err = hipMalloc((void**)&d_inBuf, NTRACKS * BUFSIZE * sizeof(float));
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to allocate d_inBuf (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMalloc((void**)&d_outBuf, NTRACKS * BUFSIZE * sizeof(float));
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to allocate d_outBuf (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMalloc((void**)&d_irBuf, NTRACKS * irLen * sizeof(float));
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to allocate d_irBuf (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Init host vectors with random floats
		// h_inBuf is copied to the device each iteration.
		for (int i = 0; i < NTRACKS * BUFSIZE; i++) {
			h_inBuf[i] = rand() / (float)RAND_MAX;
		}
		// Copy h_irBuf to device here and bind to texture once, as it's constant
		for (int i = 0; i < NTRACKS * irLen; i++) {
			h_irBuf[i] = rand() / (float)RAND_MAX;
		}
		auto channelDesc = hipCreateChannelDesc<float>();
		size_t spitch = irLen * sizeof(float);
		hipMallocArray(&cuArrayIRs, &channelDesc, NTRACKS, irLen);
		hipMemcpy2DToArray(cuArrayIRs, 0, 0, h_irBuf, spitch, NTRACKS, irLen * sizeof(float), hipMemcpyHostToDevice);
		struct hipResourceDesc rDesc;
		memset(&rDesc, 0, sizeof(rDesc));
		rDesc.resType = hipResourceTypeArray;
		rDesc.res.array.array = cuArrayIRs;
		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.addressMode[0] = hipAddressModeBorder;
		texDesc.addressMode[1] = hipAddressModeBorder;
		texDesc.filterMode = hipFilterModePoint;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = 0;
		err = hipCreateTextureObject(&texObjIRs, &rDesc, &texDesc, NULL);
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to create texObjIRs texture object (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
	}
	else if (whichBenchmark == "Modal") {
		SetupBenchmarkModal(&h_inBuf, &h_outBuf, &d_inBuf, &d_outBuf);
	}
	else {
		printf("Unknown benchmark (init section): %s\n", whichBenchmark.c_str());
		return 1;
	}

	// Main benchmark loop
	std::vector<float> latencies;

	if (whichBenchmark == "IO") {
		printf("Running I/O benchmark\n");
		for (int i = 0; i < NRUNS; i++) {
			auto start = std::chrono::high_resolution_clock::now();

			err = hipMemcpy(d_inBuf, h_inBuf, IOTEST_INBUFCOUNT * sizeof(float), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				fprintf(stderr,
					"Failed to copy vector input from host to device (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			// Launch the CUDA Kernel
			constexpr int threadsPerBlock = 256;
			constexpr int numElements_iotest = threadsPerBlock * 100;
			int blocksPerGrid = (numElements_iotest + threadsPerBlock - 1) / threadsPerBlock;
			DataTransferBenchmarkKernel << <blocksPerGrid, threadsPerBlock >> > (d_inBuf, d_outBuf, numElements_iotest);

			err = hipGetLastError();
			if (err != hipSuccess) {
				fprintf(stderr, "Failed to launch kernel (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			// Copy result back out
			err = hipMemcpy(h_outBuf, d_outBuf, IOTEST_OUTBUFCOUNT * sizeof(float), hipMemcpyDeviceToHost);

			if (err != hipSuccess) {
				fprintf(stderr,
					"Failed to copy d_outBuf from device to host (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			auto end = std::chrono::high_resolution_clock::now();
			// Compute the duration in milliseconds
			std::chrono::duration<float, std::milli> duration = end - start;
			latencies.push_back(duration.count());
			cout << "Duration: " << duration.count() << "ms" << endl;
			if (ENABLE_DAWSIM_SLEEP) {
				// Sleep for buffer interarrival time minus duration.
				std::this_thread::sleep_for(std::chrono::milliseconds(SLEEP_MS) - duration);
			}
			if (ENABLE_DAWSIM_SPIN) {
				while (std::chrono::high_resolution_clock::now() - start < std::chrono::milliseconds(SLEEP_MS)) {
					// Spin
				}
			}
		}
		printVectorStats(latencies);
		writeVectorToFile(latencies, OUTFILE);

		// Cleanup
		hipFree(d_inBuf); hipFree(d_outBuf);
		free(h_inBuf); free(h_outBuf);
	}
	else if (whichBenchmark == "Gain") {
		hipMalloc((void**)&d_stats, NTRACKS * (2) * sizeof(float));
		printf("Running Gain+Stats benchmark\n");
		for (int i = 0; i < NRUNS; i++) {
			auto start = std::chrono::high_resolution_clock::now();

			err = hipMemcpy(d_inBuf, h_inBuf, NTRACKS * BUFSIZE * sizeof(float), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				fprintf(stderr,
					"Failed to copy vector input from host to device (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			// Launch the CUDA Kernel
			constexpr int threadsPerBlock = NTRACKS;
			constexpr int numElements_iotest = NTRACKS * BUFSIZE;
			int blocksPerGrid = (numElements_iotest + threadsPerBlock - 1) / threadsPerBlock;
			GainStatsKernel << <blocksPerGrid, threadsPerBlock >> > (d_inBuf, d_outBuf, d_stats, numElements_iotest);

			err = hipGetLastError();
			if (err != hipSuccess) {
				fprintf(stderr, "Failed to launch kernel(error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			// Copy result back out
			err = hipMemcpy(h_outBuf, d_outBuf, NTRACKS * BUFSIZE * sizeof(float), hipMemcpyDeviceToHost);

			if (err != hipSuccess) {
				fprintf(stderr,
					"Failed to copy vector C from device to host (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE); // TODO and stats
			}
			// TODO: validate (other benchmarks -- or this one if we want to test data transfer internally).

			auto end = std::chrono::high_resolution_clock::now();
			// Compute the duration in milliseconds
			std::chrono::duration<float, std::milli> duration = end - start;
			latencies.push_back(duration.count());
			cout << "Duration: " << duration.count() << "ms" << endl;
		}
		printVectorStats(latencies);
		writeVectorToFile(latencies, OUTFILE);

		// Free host and device global memory. Not error checking since we're writing data and exit.
		hipFree(d_inBuf); hipFree(d_outBuf);
		free(h_inBuf); free(h_outBuf);
	}
	else if (whichBenchmark == "Conv1D_Beta") {
		// CLEANUP: move this to its own file.
		printf("Running Conv1D benchmark\n");
		for (int i = 0; i < NRUNS; i++) {
			auto start = std::chrono::high_resolution_clock::now();

			err = hipMemcpy(d_inBuf, h_inBuf, NTRACKS * BUFSIZE * sizeof(float), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				fprintf(stderr,
					"Failed to copy vector input from host to device (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			// Launch the CUDA Kernel
			constexpr int threadsPerBlock = 32;
			constexpr int numElements = NTRACKS;
			int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
			Conv1DTextureMemoryImplKernel << <blocksPerGrid, threadsPerBlock >> > (d_inBuf, d_outBuf, texObjIRs, irLen);

			err = hipGetLastError();
			if (err != hipSuccess) {
				fprintf(stderr, "Failed to launch Conv1D kernel (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			// Copy result back out
			err = hipMemcpy(h_outBuf, d_outBuf, NTRACKS * BUFSIZE * sizeof(float), hipMemcpyDeviceToHost);

			if (err != hipSuccess) {
				fprintf(stderr,
					"Failed to copy convolved audio buffer from device to host (error code %s)!\n",
					hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
			auto end = std::chrono::high_resolution_clock::now();
			// Compute the duration in milliseconds
			std::chrono::duration<float, std::milli> duration = end - start;
			latencies.push_back(duration.count());
			cout << "Duration: " << duration.count() << "ms" << endl;
		}
		printVectorStats(latencies);
		writeVectorToFile(latencies, OUTFILE);

		hipFree(d_inBuf); hipFree(d_outBuf); hipFree(d_irBuf);
		free(h_inBuf); free(h_outBuf); free(h_irBuf);
		hipDestroyTextureObject(texObjIRs);
		hipFreeArray(cuArrayIRs);
	}
	else if (whichBenchmark == "Modal") {
		RunBenchmarkModal(&d_inBuf, &h_inBuf, &d_outBuf, &h_outBuf, kNumModes, kNumModeParams, latencies);
	}
	else if (whichBenchmark == "RndMem") {
		RunBenchmarkRndMem(&d_playheads, &h_playheads, &d_sampleMem, &d_outBuf, &h_outBuf,
			latencies,
			playheadsStart, playheadsEnd);
	}
	else {
		printf("Unknown benchmark (run section): %s\n", whichBenchmark.c_str());
		return 1;
	}

	printf("Done\n");
	return 0;
}
